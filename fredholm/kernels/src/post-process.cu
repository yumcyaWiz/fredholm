#include "hip/hip_runtime.h"
#include "kernels/post-process.h"

void __host__ post_process_launch(const float4* beauty_in,
                                  const float4* denoised_in, int width,
                                  int height, float ISO, float4* beauty_out,
                                  float4* denoised_out)
{
  const dim3 threads_per_block(16, 16);
  const dim3 blocks(max(width / threads_per_block.x, 1),
                    max(height / threads_per_block.y, 1));
  post_process_kernel<<<blocks, threads_per_block>>>(
      beauty_in, denoised_in, width, height, ISO, beauty_out, denoised_out);
}

__global__ void post_process_kernel(const float4* beauty_in,
                                    const float4* denoised_in, int width,
                                    int height, float ISO, float4* beauty_out,
                                    float4* denoised_out)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height) return;
  const int image_idx = i + width * j;

  float3 color = make_float3(beauty_in[image_idx]);

  // beauty
  const float EV100 = compute_EV100(1.0f, 1.0f, ISO);
  const float exposure = convert_EV100_to_exposure(EV100);
  color *= exposure;
  // color = aces_tone_mapping(color);
  color = uchimura(color);
  color = linear_to_srgb(color);
  beauty_out[image_idx] = make_float4(color, 1.0f);

  // denoised
  color = make_float3(denoised_in[image_idx]);
  color *= exposure;
  // color = aces_tone_mapping(color);
  color = uchimura(color);
  color = linear_to_srgb(color);
  denoised_out[image_idx] = make_float4(color, 1.0f);
}