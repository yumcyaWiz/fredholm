#include "hip/hip_runtime.h"
#include "cwl/util.h"
#include "kernels/post-process.h"
#include "sutil/vec_math.h"

void __host__ post_process_kernel_launch(
    const float4* beauty_in, const float4* denoised_in,
    float4* beauty_high_luminance, float4* denoised_high_luminance,
    float4* beauty_temp, float4* denoised_temp, int width, int height,
    float bloom_threshold, float bloom_sigma, float ISO, float4* beauty_out,
    float4* denoised_out)
{
  const dim3 threads_per_block(16, 16);
  const dim3 blocks(max(width / threads_per_block.x, 1),
                    max(height / threads_per_block.y, 1));

  // extract high luminance pixels
  bloom_kernel_0<<<blocks, threads_per_block>>>(
      beauty_in, denoised_in, width, height, bloom_threshold,
      beauty_high_luminance, denoised_high_luminance);
  CUDA_SYNC_CHECK();

  // gaussian blur
  bloom_kernel_1<<<blocks, threads_per_block>>>(
      beauty_in, denoised_in, beauty_high_luminance, denoised_high_luminance,
      width, height, bloom_sigma, beauty_temp, denoised_temp);
  CUDA_SYNC_CHECK();

  // tone mapping
  tone_mapping_kernel<<<blocks, threads_per_block>>>(
      beauty_temp, denoised_temp, width, height, ISO, beauty_out, denoised_out);
}

void __host__ tone_mapping_kernel_launch(const float4* beauty_in,
                                         const float4* denoised_in, int width,
                                         int height, float ISO,
                                         float4* beauty_out,
                                         float4* denoised_out)
{
  const dim3 threads_per_block(16, 16);
  const dim3 blocks(max(width / threads_per_block.x, 1),
                    max(height / threads_per_block.y, 1));
  tone_mapping_kernel<<<blocks, threads_per_block>>>(
      beauty_in, denoised_in, width, height, ISO, beauty_out, denoised_out);
}

__global__ void bloom_kernel_0(const float4* beauty_in,
                               const float4* denoised_in, int width, int height,
                               float bloom_threshold, float4* beauty_out,
                               float4* denoised_out)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height) return;
  const int image_idx = i + width * j;

  const float4 beauty = beauty_in[image_idx];
  const float4 denoised = denoised_in[image_idx];

  const float beauty_luminance = rgb_to_luminance(make_float3(beauty));
  const float denoised_luminance = rgb_to_luminance(make_float3(denoised));

  beauty_out[image_idx] =
      beauty_luminance > bloom_threshold ? beauty : make_float4(0.0f);
  denoised_out[image_idx] =
      denoised_luminance > bloom_threshold ? denoised : make_float4(0.0f);
}

__global__ void bloom_kernel_1(const float4* beauty_in,
                               const float4* denoised_in,
                               const float4* beauty_high_luminance,
                               const float4* denoised_high_luminance, int width,
                               int height, float bloom_sigma,
                               float4* beauty_out, float4* denoised_out)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height) return;
  const int image_idx = i + width * j;

  const float4 b0 = beauty_in[image_idx];
  const float4 d0 = denoised_in[image_idx];

  const int K = 16;
  const float sigma = 1.0f;

  float4 b_sum = make_float4(0.0f);
  float4 d_sum = make_float4(0.0f);
  float w_sum = 0.0f;
  for (int v = -K; v <= K; ++v) {
    for (int u = -K; u <= K; ++u) {
      const int x = clamp(i + u, 0, width - 1);
      const int y = clamp(j + v, 0, height - 1);

      const float4 b1 = beauty_high_luminance[x + width * y];
      const float4 d1 = denoised_high_luminance[x + width * y];

      const float dist2 = u * u + v * v;
      const float h = expf(-dist2 / (2.0f * bloom_sigma));

      b_sum += h * b1;
      d_sum += h * d1;
      w_sum += h;
    }
  }

  beauty_out[image_idx] = b0 + b_sum / w_sum;
  denoised_out[image_idx] = d0 + d_sum / w_sum;
}

__global__ void tone_mapping_kernel(const float4* beauty_in,
                                    const float4* denoised_in, int width,
                                    int height, float ISO, float4* beauty_out,
                                    float4* denoised_out)
{
  const int i = blockIdx.x * blockDim.x + threadIdx.x;
  const int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height) return;
  const int image_idx = i + width * j;

  // chromatic aberration
  const float2 uv = make_float2(static_cast<float>(i) / width,
                                static_cast<float>(j) / height);
  const float2 d = (uv - make_float2(0.5f)) * 0.0000075f * 0.2f;

  const float2 uv_r =
      clamp(uv - 0.0f * d, make_float2(0.0f), make_float2(1.0f));
  const float2 uv_g =
      clamp(uv - 1.0f * d, make_float2(0.0f), make_float2(1.0f));
  const float2 uv_b =
      clamp(uv - 2.0f * d, make_float2(0.0f), make_float2(1.0f));

  const int image_idx_r = uv_r.x * width + width * uv_r.y * height;
  const int image_idx_g = uv_g.x * width + width * uv_g.y * height;
  const int image_idx_b = uv_b.x * width + width * uv_b.y * height;

  // beauty
  float3 color = make_float3(beauty_in[image_idx_r].x, beauty_in[image_idx_g].y,
                             beauty_in[image_idx_b].z);
  const float EV100 = compute_EV100(1.0f, 1.0f, ISO);
  const float exposure = convert_EV100_to_exposure(EV100);
  color *= exposure;
  // color = aces_tone_mapping(color);
  color = uchimura(color);
  color = linear_to_srgb(color);
  beauty_out[image_idx] = make_float4(color, 1.0f);

  // denoised
  color = make_float3(denoised_in[image_idx_r].x, denoised_in[image_idx_g].y,
                      denoised_in[image_idx_b].z);
  color *= exposure;
  // color = aces_tone_mapping(color);
  color = uchimura(color);
  color = linear_to_srgb(color);
  denoised_out[image_idx] = make_float4(color, 1.0f);
}