#include "hip/hip_runtime.h"
#include <optix.h>

#include "math.cu"
#include "sampling.cu"
#include "shared.h"
#include "sutil/vec_math.h"

#define RAY_EPS 0.001f

extern "C" {
__constant__ LaunchParams params;
}

enum class RayType : unsigned int {
  RAY_TYPE_RADIANCE = 0,
  RAY_TYPE_SHADOW = 1,
  RAY_TYPE_COUNT
};

struct RadiancePayload {
  float3 origin;
  float3 direction;

  float3 throughput = make_float3(1);
  float3 radiance = make_float3(0);

  RNGState rng;

  bool done = false;
};

// upper-32bit + lower-32bit -> 64bit
static __forceinline__ __device__ void* unpack_ptr(unsigned int i0,
                                                   unsigned int i1)
{
  const unsigned long long uptr =
      static_cast<unsigned long long>(i0) << 32 | i1;
  void* ptr = reinterpret_cast<void*>(uptr);
  return ptr;
}

// 64bit -> upper-32bit + lower-32bit
static __forceinline__ __device__ void pack_ptr(void* ptr, unsigned int& i0,
                                                unsigned int& i1)
{
  const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
  i0 = uptr >> 32;
  i1 = uptr & 0x00000000ffffffff;
}

// u0, u1 is upper-32bit, lower-32bit of ptr of Payload
template <typename Payload>
static __forceinline__ __device__ Payload* get_payload_ptr()
{
  const unsigned int u0 = optixGetPayload_0();
  const unsigned int u1 = optixGetPayload_1();
  return reinterpret_cast<Payload*>(unpack_ptr(u0, u1));
}

// trace radiance ray
static __forceinline__ __device__ void trace_radiance(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    RadiancePayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE), u0, u1);
}

static __forceinline__ __device__ bool has_emission(const Material& material)
{
  return (material.emission_color.x > 0 || material.emission_color.y > 0 ||
          material.emission_color.z > 0);
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    const float2& uv, float3& origin, float3& direction)
{
  const float3 p_sensor =
      params.cam_origin + uv.x * params.cam_right + uv.y * params.cam_up;
  const float3 p_pinhole = params.cam_origin + params.cam_forward;

  origin = params.cam_origin;
  direction = normalize(p_pinhole - p_sensor);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();

  float3 radiance = make_float3(0);

  // warm up rng
  // TODO: use some hash function to set more nice seed
  RadiancePayload payload;
  payload.rng.state = idx.x + params.width * idx.y;
  for (int i = 0; i < 10; ++i) { frandom(payload.rng); }

  for (int spp = 0; spp < params.n_samples; ++spp) {
    // generate initial ray from camera
    const float2 uv =
        make_float2((2.0f * (idx.x + frandom(payload.rng)) - dim.x) / dim.x,
                    (2.0f * (idx.y + frandom(payload.rng)) - dim.y) / dim.y);
    sample_ray_pinhole_camera(uv, payload.origin, payload.direction);

    // start ray tracing from the camera
    payload.radiance = make_float3(0);
    payload.throughput = make_float3(1);
    payload.done = false;
    for (int depth = 0; depth < params.max_depth; ++depth) {
      trace_radiance(params.gas_handle, payload.origin, payload.direction, 0.0f,
                     1e9f, &payload);

      if (payload.done) { break; }
    }

    // accumulate contribution
    radiance += payload.radiance;
  }

  // take average
  radiance /= params.n_samples;

  // gamma correction
  radiance.x = pow(radiance.x, 1.0f / 2.2f);
  radiance.y = pow(radiance.y, 1.0f / 2.2f);
  radiance.z = pow(radiance.z, 1.0f / 2.2f);

  // write radiance to frame buffer
  params.framebuffer[idx.x + params.width * idx.y] =
      make_float4(radiance, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
  const MissSbtRecordData* sbt =
      reinterpret_cast<MissSbtRecordData*>(optixGetSbtDataPointer());

  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  payload->radiance += payload->throughput * sbt->bg_color;
  payload->done = true;
}

extern "C" __global__ void __miss__shadow() {}

extern "C" __global__ void __closesthit__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());

  // Le
  if (has_emission(sbt->material)) {
    payload->radiance += payload->throughput * sbt->material.emission_color;
    payload->done = true;
    return;
  }

  // compute face normal
  // TODO: remove this calculation, store normals(ptr) in SBT
  const int prim_idx = optixGetPrimitiveIndex();
  const float3 v0 = sbt->vertices[3 * prim_idx + 0];
  const float3 v1 = sbt->vertices[3 * prim_idx + 1];
  const float3 v2 = sbt->vertices[3 * prim_idx + 2];
  const float3 n = normalize(cross(v1 - v0, v2 - v0));

  // compute tangent space basis
  float3 t, b;
  orthonormal_basis(n, t, b);

  // sample next ray direction
  const float3 wi = sample_cosine_weighted_hemisphere(frandom(payload->rng),
                                                      frandom(payload->rng));
  const float3 wi_world = local_to_world(wi, t, n, b);

  // update payload
  payload->throughput *= sbt->material.base_color;

  // advance ray
  payload->origin = optixGetWorldRayOrigin() +
                    optixGetRayTmax() * optixGetWorldRayDirection();
  payload->origin += RAY_EPS * n;
  payload->direction = wi_world;
}

extern "C" __global__ void __closesthit__shadow() {}