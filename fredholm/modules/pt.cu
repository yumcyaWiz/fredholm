#include "hip/hip_runtime.h"
#include <optix.h>

#include "bsdf.cu"
#include "fredholm/shared.h"
#include "math.cu"
#include "sampling.cu"
#include "sutil/vec_math.h"

using namespace fredholm;

extern "C" {
__constant__ LaunchParams params;
}

struct RadiancePayload {
  float3 origin;
  float3 direction;

  float3 throughput = make_float3(1);
  float3 radiance = make_float3(0);

  SamplerState sampler;

  bool done = false;

  bool firsthit = true;
  float3 position = make_float3(0);
  float3 normal = make_float3(0);
  float depth = 0;
  float2 texcoord = make_float2(0);
  float3 albedo = make_float3(0);
};

struct ShadowPayload {
  bool visible = false;  // light visibility
};

struct LightPayload {
  float3 direction;  // ray direction

  float3 le = make_float3(0.0f);  // emission

  bool hit = false;  // hit area light?
  float3 p;          // hit position
  float3 n;          // hit normal
  float area;        // triangle area
};

// upper-32bit + lower-32bit -> 64bit
static __forceinline__ __device__ void* unpack_ptr(unsigned int i0,
                                                   unsigned int i1)
{
  const unsigned long long uptr =
      static_cast<unsigned long long>(i0) << 32 | i1;
  void* ptr = reinterpret_cast<void*>(uptr);
  return ptr;
}

// 64bit -> upper-32bit + lower-32bit
static __forceinline__ __device__ void pack_ptr(void* ptr, unsigned int& i0,
                                                unsigned int& i1)
{
  const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
  i0 = uptr >> 32;
  i1 = uptr & 0x00000000ffffffff;
}

// u0, u1 is upper-32bit, lower-32bit of ptr of Payload
template <typename Payload>
static __forceinline__ __device__ Payload* get_payload_ptr()
{
  const unsigned int u0 = optixGetPayload_0();
  const unsigned int u1 = optixGetPayload_1();
  return reinterpret_cast<Payload*>(unpack_ptr(u0, u1));
}

// trace radiance ray
static __forceinline__ __device__ void trace_radiance(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    RadiancePayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE), u0, u1);
}

static __forceinline__ __device__ void trace_shadow(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    ShadowPayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW), u0, u1);
}

static __forceinline__ __device__ void trace_light(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    LightPayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_LIGHT),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_LIGHT), u0, u1);
}

static __forceinline__ __device__ bool has_emission(const Material& material)
{
  return (material.emission_color.x > 0 || material.emission_color.y > 0 ||
          material.emission_color.z > 0);
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    const float2& uv, float3& origin, float3& direction, float& pdf)
{
  const float3 p_sensor = params.camera.origin + uv.x * params.camera.right +
                          uv.y * params.camera.up;
  const float3 p_pinhole =
      params.camera.origin + params.camera.f * params.camera.forward;

  origin = p_pinhole;
  direction = normalize(p_pinhole - p_sensor);
  pdf = 1.0f / dot(direction, params.camera.forward);
}

static __forceinline__ __device__ void fill_surface_info(
    const float3* vertices, const uint3* indices, const float3* normals,
    const float2* texcoords, const float3& ray_origin,
    const float3& ray_direction, float ray_tmax, const float2& barycentric,
    uint prim_idx, SurfaceInfo& info)
{
  info.t = ray_tmax;
  info.barycentric = barycentric;

  const uint3 idx = indices[prim_idx];
  const float3 v0 = vertices[idx.x];
  const float3 v1 = vertices[idx.y];
  const float3 v2 = vertices[idx.z];
  // surface based robust hit position, Ray Tracing Gems Chapter 6
  info.x = (1.0f - info.barycentric.x - info.barycentric.y) * v0 +
           info.barycentric.x * v1 + info.barycentric.y * v2;
  info.n_g = normalize(cross(v1 - v0, v2 - v0));

  const float3 n0 = normals[idx.x];
  const float3 n1 = normals[idx.y];
  const float3 n2 = normals[idx.z];
  info.n_s = normalize((1.0f - info.barycentric.x - info.barycentric.y) * n0 +
                       info.barycentric.x * n1 + info.barycentric.y * n2);

  const float2 tex0 = texcoords[idx.x];
  const float2 tex1 = texcoords[idx.y];
  const float2 tex2 = texcoords[idx.z];
  info.texcoord = (1.0f - info.barycentric.x - info.barycentric.y) * tex0 +
                  info.barycentric.x * tex1 + info.barycentric.y * tex2;

  // flip normal
  info.is_entering = dot(-ray_direction, info.n_g) > 0;
  info.n_s = info.is_entering ? info.n_s : -info.n_s;
  info.n_g = info.is_entering ? info.n_g : -info.n_g;

  orthonormal_basis(info.n_s, info.tangent, info.bitangent);
}

static __forceinline__ __device__ ShadingParams fill_shading_params(
    const Material& material, const SurfaceInfo& surf_info,
    const TextureHeader* textures, ShadingParams& shading_params)
{
  // diffuse
  shading_params.diffuse = material.diffuse;

  // diffuse roughness
  shading_params.diffuse_roughness = material.diffuse_roughness;

  // base color
  shading_params.base_color =
      material.base_color_texture_id >= 0
          ? make_float3(tex2D<float4>(
                textures[material.base_color_texture_id].texture_object,
                surf_info.texcoord.x, surf_info.texcoord.y))
          : material.base_color;

  // specular
  shading_params.specular = material.specular;

  // specular color
  shading_params.specular_color =
      material.specular_color_texture_id >= 0
          ? make_float3(tex2D<float4>(
                textures[material.specular_color_texture_id].texture_object,
                surf_info.texcoord.x, surf_info.texcoord.y))
          : material.specular_color;

  // specular roughness
  shading_params.specular_roughness =
      material.specular_roughness_texture_id >= 0
          ? tex2D<float4>(
                textures[material.specular_roughness_texture_id].texture_object,
                surf_info.texcoord.x, surf_info.texcoord.y)
                .x
          : material.specular_roughness;

  // metalness
  shading_params.metalness =
      material.metalness_texture_id >= 0
          ? tex2D<float4>(
                textures[material.metalness_texture_id].texture_object,
                surf_info.texcoord.x, surf_info.texcoord.y)
                .x
          : material.metalness;

  // coat
  shading_params.coat = material.coat;

  // coat roughness
  shading_params.coat_roughness = material.coat_roughness;

  // transmission
  shading_params.transmission = material.transmission;

  // transmission color
  shading_params.transmission_color = material.transmission_color;

  // subsurface
  shading_params.subsurface = material.subsurface;

  // subsurface color
  shading_params.subsurface_color = material.subsurface_color;

  // thin walled
  shading_params.thin_walled = material.thin_walled;
}

static __forceinline__ __device__ float3
sample_position_on_light(const float u, const float2& v, const float3* vertices,
                         const uint3* indices, const float3* normals,
                         float3& le, float3& n, float& pdf)
{
  // sample light
  const uint light_idx =
      clamp(static_cast<uint>(u * params.n_lights), 0u, params.n_lights - 1);
  const AreaLight& light = params.lights[light_idx];

  // sample point on the light
  const float2 barycentric = sample_triangle(v);

  const uint3 idx = light.indices;
  const float3 v0 = vertices[idx.x];
  const float3 v1 = vertices[idx.y];
  const float3 v2 = vertices[idx.z];
  const float3 n0 = normals[idx.x];
  const float3 n1 = normals[idx.y];
  const float3 n2 = normals[idx.z];

  const float3 p = (1.0f - barycentric.x - barycentric.y) * v0 +
                   barycentric.x * v1 + barycentric.y * v2;
  n = (1.0f - barycentric.x - barycentric.y) * n0 + barycentric.x * n1 +
      barycentric.y * n2;
  const float area = 0.5f * length(cross(v1 - v0, v2 - v0));

  le = light.le;
  pdf = 1.0f / (params.n_lights * area);

  return p;
}

static __forceinline__ __device__ float3
sample_position_on_directional_light(const float2& u)
{
  constexpr float DIRECTIONAL_LIGHT_DISTANCE = 1e9f;

  // sample point on disk
  const float2 p_disk = sample_concentric_disk(u);

  // compute world space position
  const float disk_radius =
      DIRECTIONAL_LIGHT_DISTANCE *
      tanf(deg_to_rad(0.5f * params.directional_light->angle));
  float3 t, b;
  orthonormal_basis(params.directional_light->dir, t, b);
  const float3 p = DIRECTIONAL_LIGHT_DISTANCE * params.directional_light->dir +
                   disk_radius * (t * p_disk.x + b * p_disk.y);

  return p;
}

static __forceinline__ __device__ float3 fetch_ibl(const float3& v)
{
  const float2 thphi = cartesian_to_spherical(v);
  return make_float3(
      tex2D<float4>(params.ibl, thphi.y / (2.0f * M_PIf), thphi.x / M_PIf));
}

// power heuristics
static __forceinline__ __device__ float compute_mis_weight(float pdf0,
                                                           float pdf1)
{
  return (pdf0) / (pdf0 + pdf1);
}

// TODO: need more nice way to suppress firefly
static __forceinline__ __device__ float3 regularize_weight(const float3& weight)
{
  return clamp(weight, make_float3(0.0f), make_float3(1.0f));
}

static __forceinline__ __device__ void init_sampler_state(
    unsigned int image_idx, unsigned int n_spp, SamplerState& state)
{
  state.pcg_state.state =
      xxhash32(image_idx + n_spp * params.width * params.height);
  state.pcg_state.inc = xxhash32(params.seed);

  state.sobol_state.index = image_idx + n_spp * params.width * params.height;
  state.sobol_state.dimension = 1;
  state.sobol_state.seed = xxhash32(params.seed);

  state.cmj_state.image_idx = image_idx;
  state.cmj_state.depth = 0;
  state.cmj_state.n_spp = n_spp;
  state.cmj_state.scramble = xxhash32(params.seed);
}

// Ray Tracing Gems Chapter 6
static __forceinline__ __device__ float3 ray_origin_offset(const float3& p,
                                                           const float3& n)
{
  constexpr float origin = 1.0f / 32.0f;
  constexpr float float_scale = 1.0f / 65536.0f;
  constexpr float int_scale = 256.0f;
  const int3 of_i = make_int3(int_scale * n);
  const float3 p_i = make_float3(
      __int_as_float(__float_as_int(p.x) + ((p.x < 0) ? -of_i.x : of_i.x)),
      __int_as_float(__float_as_int(p.y) + ((p.y < 0) ? -of_i.y : of_i.y)),
      __int_as_float(__float_as_int(p.z) + ((p.z < 0) ? -of_i.z : of_i.z)));
  return make_float3(fabsf(p.x) < origin ? p.x + float_scale * n.x : p_i.x,
                     fabsf(p.y) < origin ? p.y + float_scale * n.y : p_i.y,
                     fabsf(p.z) < origin ? p.z + float_scale * n.z : p_i.z);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();
  const uint image_idx = idx.x + params.width * idx.y;
  uint n_spp = params.sample_count[image_idx];

  float3 beauty = make_float3(params.render_layer.beauty[image_idx]);
  float3 position = make_float3(params.render_layer.position[image_idx]);
  float3 normal = make_float3(params.render_layer.normal[image_idx]);
  float depth = params.render_layer.depth[image_idx];
  float2 texcoord = make_float2(params.render_layer.texcoord[image_idx]);
  float3 albedo = make_float3(params.render_layer.albedo[image_idx]);

  RadiancePayload payload;
  for (int spp = 0; spp < params.n_samples; ++spp) {
    // initialize sampler
    init_sampler_state(image_idx, n_spp, payload.sampler);

    // generate initial ray from camera
    const float2 u = sample_2d(payload.sampler);
    float2 uv = make_float2((2.0f * (idx.x + u.x) - dim.x) / dim.y,
                            (2.0f * (idx.y + u.y) - dim.y) / dim.y);
    // flip x
    uv.x = -uv.x;
    float camera_pdf;
    sample_ray_pinhole_camera(uv, payload.origin, payload.direction,
                              camera_pdf);

    // start ray tracing from the camera
    payload.radiance = make_float3(0);
    payload.throughput =
        make_float3(dot(payload.direction, params.camera.forward) / camera_pdf);
    payload.done = false;
    for (int ray_depth = 0; ray_depth < params.max_depth; ++ray_depth) {
      // russian roulette
      const float russian_roulette_prob =
          ray_depth == 0
              ? 1.0f
              : clamp(rgb_to_luminance(payload.throughput), 0.0f, 1.0f);
      if (sample_1d(payload.sampler) >= russian_roulette_prob) { break; }
      payload.throughput /= russian_roulette_prob;

      // trace ray and update payloads
      trace_radiance(params.ias_handle, payload.origin, payload.direction, 0.0f,
                     1e9f, &payload);

      // throughput nan check
      if (isnan(payload.throughput) || isinf(payload.throughput)) { break; }

      if (payload.done) { break; }
    }

    // radiance nan check
    float3 radiance = make_float3(0.0f);
    if (!isnan(payload.radiance) && !isinf(payload.radiance)) {
      radiance = payload.radiance;
    }

    // take streaming average
    const float coef = 1.0f / (n_spp + 1.0f);
    beauty = coef * (n_spp * beauty + radiance);
    position = coef * (n_spp * position + payload.position);
    normal = coef * (n_spp * normal + payload.normal);
    depth = coef * (n_spp * depth + payload.depth);
    texcoord = coef * (n_spp * texcoord + payload.texcoord);
    albedo = coef * (n_spp * albedo + payload.albedo);

    n_spp++;
  }

  // update total number of samples
  params.sample_count[image_idx] = n_spp;

  // write results in render layers
  params.render_layer.beauty[image_idx] = make_float4(beauty, 1.0f);
  params.render_layer.position[image_idx] = make_float4(position, 1.0f);
  params.render_layer.normal[image_idx] = make_float4(normal, 1.0f);
  params.render_layer.depth[image_idx] = depth;
  params.render_layer.texcoord[image_idx] = make_float4(texcoord, 0.0f, 1.0f);
  params.render_layer.albedo[image_idx] = make_float4(albedo, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();

  // firsthit light case
  if (payload->firsthit) {
    float3 le;
    if (params.ibl) {
      le = fetch_ibl(payload->direction);
    } else {
      le = params.bg_color;
    }

    payload->radiance += payload->throughput * le;
  }

  payload->done = true;
}

extern "C" __global__ void __miss__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visible = true;
}

extern "C" __global__ void __miss__light()
{
  LightPayload* payload = get_payload_ptr<LightPayload>();
  payload->hit = false;

  if (params.ibl) {
    payload->le = fetch_ibl(payload->direction);
  } else {
    payload->le = params.bg_color;
  }
}

extern "C" __global__ void __anyhit__radiance()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(
            params.textures[material.base_color_texture_id].texture_object,
            texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id].texture_object,
                      texcoord.x, texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __anyhit__shadow()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(
            params.textures[material.base_color_texture_id].texture_object,
            texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id].texture_object,
                      texcoord.x, texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __anyhit__light()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(
            params.textures[material.base_color_texture_id].texture_object,
            texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id].texture_object,
                      texcoord.x, texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __closesthit__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();

  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  const float3 ray_origin = optixGetWorldRayOrigin();
  const float3 ray_direction = optixGetWorldRayDirection();
  const float ray_tmax = optixGetRayTmax();
  const float2 barycentric = optixGetTriangleBarycentrics();

  SurfaceInfo surf_info;
  fill_surface_info(sbt->vertices, sbt->indices, sbt->normals, sbt->texcoords,
                    ray_origin, ray_direction, ray_tmax, barycentric, prim_idx,
                    surf_info);

  ShadingParams shading_params;
  fill_shading_params(material, surf_info, params.textures, shading_params);

  float3 tangent = surf_info.tangent;
  float3 normal = surf_info.n_s;
  float3 bitangent = surf_info.bitangent;

  // bump mapping(with height map)
  if (material.heightmap_texture_id >= 0) {
    const TextureHeader& heightmap =
        params.textures[material.heightmap_texture_id];
    const float du = 1.0f / heightmap.size.x;
    const float dv = 1.0f / heightmap.size.y;
    const float v = tex2D<float4>(heightmap.texture_object,
                                  surf_info.texcoord.x, surf_info.texcoord.y)
                        .x;
    const float dfdu =
        (tex2D<float4>(heightmap.texture_object, surf_info.texcoord.x + du,
                       surf_info.texcoord.y)
             .x -
         v);
    const float dfdv =
        (tex2D<float4>(heightmap.texture_object, surf_info.texcoord.x,
                       surf_info.texcoord.y + dv)
             .x -
         v);
    tangent = normalize(surf_info.tangent + dfdu * surf_info.n_s);
    bitangent = normalize(surf_info.bitangent + dfdv * surf_info.n_s);
    normal = normalize(cross(tangent, bitangent));
  }

  // normal mapping
  if (material.normalmap_texture_id >= 0) {
    float3 value = make_float3(tex2D<float4>(
        params.textures[material.normalmap_texture_id].texture_object,
        surf_info.texcoord.x, surf_info.texcoord.y));
    value = 2.0f * value - 1.0f;
    normal = normalize(local_to_world(value, surf_info.tangent,
                                      surf_info.bitangent, surf_info.n_s));
    orthonormal_basis(normal, tangent, bitangent);
  }

  // fill position, normal, depth, albedo
  if (payload->firsthit) {
    payload->position = surf_info.x;
    payload->normal = normal;
    payload->depth = surf_info.t;
    payload->texcoord = surf_info.texcoord;
    payload->albedo = shading_params.base_color;
    payload->firsthit = false;

    // first hit light case
    if (has_emission(material)) {
      payload->radiance += payload->throughput * material.emission_color;
      payload->done = true;
      return;
    }
  }

  // init BSDF
  const float3 wo = world_to_local(-ray_direction, tangent, normal, bitangent);
  const BSDF bsdf = BSDF(wo, shading_params, surf_info.is_entering);

  // light sampling
  {
    const float3 shadow_ray_origin =
        ray_origin_offset(surf_info.x, surf_info.n_g);

    // directional light
    if (params.directional_light) {
      const float3 shadow_ray_direction = normalize(
          sample_position_on_directional_light(sample_2d(payload->sampler)) -
          shadow_ray_origin);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, 1e9f, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 wi =
            world_to_local(shadow_ray_direction, tangent, normal, bitangent);
        const float3 f = bsdf.eval(wo, wi);
        const float pdf = 1.0f;
        const float pdf_bsdf = bsdf.eval_pdf(wo, wi);
        const float mis_weight = compute_mis_weight(pdf, pdf_bsdf);
        const float3 weight = regularize_weight(
            payload->throughput * mis_weight * f * abs_cos_theta(wi) / pdf);
        const float3 le = params.directional_light->le;
        payload->radiance += weight * le;
      }
    }

    // sky
    if (params.ibl) {
      // TODO: implement IBL importance sampling
      const float3 wi =
          sample_cosine_weighted_hemisphere(sample_2d(payload->sampler));
      const float3 shadow_ray_direction =
          local_to_world(wi, tangent, normal, bitangent);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, 1e9f, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 f = bsdf.eval(wo, wi);
        const float pdf = abs_cos_theta(wi) / M_PIf;
        const float pdf_bsdf = bsdf.eval_pdf(wo, wi);
        const float mis_weight = compute_mis_weight(pdf, pdf_bsdf);
        const float3 weight = regularize_weight(
            payload->throughput * mis_weight * f * abs_cos_theta(wi) / pdf);
        const float3 le = fetch_ibl(shadow_ray_direction);
        payload->radiance += weight * le;
      }
    } else {
      const float3 wi =
          sample_cosine_weighted_hemisphere(sample_2d(payload->sampler));
      const float3 shadow_ray_direction =
          local_to_world(wi, tangent, normal, bitangent);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, 1e9f, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 f = bsdf.eval(wo, wi);
        const float pdf = abs_cos_theta(wi) / M_PIf;
        const float pdf_bsdf = bsdf.eval_pdf(wo, wi);
        const float mis_weight = compute_mis_weight(pdf, pdf_bsdf);
        const float3 weight = regularize_weight(
            payload->throughput * mis_weight * f * abs_cos_theta(wi) / pdf);
        const float3 le = params.bg_color;
        payload->radiance += weight * le;
      }
    }

    // area light
    if (params.n_lights > 0) {
      float3 le, n;
      float pdf_area;
      const float3 p = sample_position_on_light(
          sample_1d(payload->sampler), sample_2d(payload->sampler),
          sbt->vertices, sbt->indices, sbt->normals, le, n, pdf_area);

      const float3 shadow_ray_direction = normalize(p - shadow_ray_origin);
      const float r = length(p - shadow_ray_origin);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, r, &shadow_payload);

      if (shadow_payload.visible && dot(-shadow_ray_direction, n) > 0.0f) {
        const float3 wi =
            world_to_local(shadow_ray_direction, tangent, normal, bitangent);
        const float3 f = bsdf.eval(wo, wi);
        float pdf = r * r / fabs(dot(-shadow_ray_direction, n)) * pdf_area;

        const float pdf_bsdf = bsdf.eval_pdf(wo, wi);
        const float mis_weight = compute_mis_weight(pdf, pdf_bsdf);
        const float3 weight = regularize_weight(
            payload->throughput * mis_weight * f * abs_cos_theta(wi) / pdf);
        payload->radiance += weight * le;
      }
    }
  }

  // BSDF sampling
  {
    float3 f;
    float pdf;
    const float3 wi = bsdf.sample(wo, sample_1d(payload->sampler),
                                  sample_2d(payload->sampler), f, pdf);

    const float3 light_ray_direction =
        local_to_world(wi, tangent, normal, bitangent);
    const bool is_transmitted = dot(light_ray_direction, surf_info.n_g) < 0;
    const float3 light_ray_origin = ray_origin_offset(
        surf_info.x, is_transmitted ? -surf_info.n_g : surf_info.n_g);

    LightPayload light_payload;
    light_payload.direction = light_ray_direction;
    trace_light(params.ias_handle, light_ray_origin, light_ray_direction, 0.0f,
                1e9f, &light_payload);

    float pdf_light;
    if (light_payload.hit) {
      const float r2 = dot(light_payload.p - light_ray_origin,
                           light_payload.p - light_ray_origin);
      const float pdf_area = 1.0f / (params.n_lights * light_payload.area);
      pdf_light =
          r2 / fabs(dot(-light_ray_direction, light_payload.n)) * pdf_area;
    } else {
      pdf_light = abs_cos_theta(wi) / M_PIf;
    }

    const float mis_weight = compute_mis_weight(pdf, pdf_light);
    const float3 weight = regularize_weight(payload->throughput * mis_weight *
                                            f * abs_cos_theta(wi) / pdf);
    payload->radiance += weight * light_payload.le;
  }

  // generate next ray direction
  {
    float3 f;
    float pdf;
    const float3 wi = bsdf.sample(wo, sample_1d(payload->sampler),
                                  sample_2d(payload->sampler), f, pdf);
    const float3 wi_world = local_to_world(wi, tangent, normal, bitangent);

    // update throughput
    payload->throughput *= f * abs_cos_theta(wi) / pdf;

    // advance ray
    const bool is_transmitted = dot(wi_world, surf_info.n_g) < 0;
    payload->origin = ray_origin_offset(
        surf_info.x, is_transmitted ? -surf_info.n_g : surf_info.n_g);
    payload->direction = wi_world;
  }
}

extern "C" __global__ void __closesthit__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visible = false;
}

extern "C" __global__ void __closesthit__light()
{
  LightPayload* payload = get_payload_ptr<LightPayload>();

  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  const uint3 idx = sbt->indices[prim_idx];
  const float3 v0 = sbt->vertices[idx.x];
  const float3 v1 = sbt->vertices[idx.y];
  const float3 v2 = sbt->vertices[idx.z];
  const float3 n0 = sbt->normals[idx.x];
  const float3 n1 = sbt->normals[idx.y];
  const float3 n2 = sbt->normals[idx.z];

  const float2 barycentric = optixGetTriangleBarycentrics();
  const float3 p = (1.0f - barycentric.x - barycentric.y) * v0 +
                   barycentric.x * v1 + barycentric.y * v2;
  const float3 n = (1.0f - barycentric.x - barycentric.y) * n0 +
                   barycentric.x * n1 + barycentric.y * n2;

  if (has_emission(material) && dot(-payload->direction, n) > 0.0f) {
    payload->hit = true;
    payload->le = material.emission_color;
    payload->p = p;
    payload->n = n;
    payload->area = 0.5f * length(cross(v1 - v0, v2 - v0));
  } else {
    payload->hit = false;
    payload->le = make_float3(0.0f);
  }
}