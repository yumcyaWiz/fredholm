#include "hip/hip_runtime.h"
#include <optix.h>

#include "bsdf.cu"
#include "fredholm/shared.h"
#include "math.cu"
#include "sampling.cu"
#include "sutil/vec_math.h"

#define RAY_EPS 0.001f

using namespace fredholm;

extern "C" {
__constant__ LaunchParams params;
}

struct RadiancePayload {
  float3 origin;
  float3 direction;

  float3 throughput = make_float3(1);
  float3 radiance = make_float3(0);

  RNGState rng;

  bool done = false;

  bool firsthit = true;
  float3 position = make_float3(0);
  float3 normal = make_float3(0);
  float depth = 0;
  float2 texcoord = make_float2(0);
  float3 albedo = make_float3(0);
};

struct ShadowPayload {
  bool visible = false;  // light visibility
};

struct LightPayload {
  float3 direction;
  float3 le = make_float3(0.0f);
};

// upper-32bit + lower-32bit -> 64bit
static __forceinline__ __device__ void* unpack_ptr(unsigned int i0,
                                                   unsigned int i1)
{
  const unsigned long long uptr =
      static_cast<unsigned long long>(i0) << 32 | i1;
  void* ptr = reinterpret_cast<void*>(uptr);
  return ptr;
}

// 64bit -> upper-32bit + lower-32bit
static __forceinline__ __device__ void pack_ptr(void* ptr, unsigned int& i0,
                                                unsigned int& i1)
{
  const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
  i0 = uptr >> 32;
  i1 = uptr & 0x00000000ffffffff;
}

// u0, u1 is upper-32bit, lower-32bit of ptr of Payload
template <typename Payload>
static __forceinline__ __device__ Payload* get_payload_ptr()
{
  const unsigned int u0 = optixGetPayload_0();
  const unsigned int u1 = optixGetPayload_1();
  return reinterpret_cast<Payload*>(unpack_ptr(u0, u1));
}

// trace radiance ray
static __forceinline__ __device__ void trace_radiance(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    RadiancePayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE), u0, u1);
}

static __forceinline__ __device__ void trace_shadow(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    ShadowPayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW), u0, u1);
}

static __forceinline__ __device__ void trace_light(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    LightPayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_LIGHT),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_LIGHT), u0, u1);
}

static __forceinline__ __device__ bool has_emission(const Material& material)
{
  return (material.emission_color.x > 0 || material.emission_color.y > 0 ||
          material.emission_color.z > 0);
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    const float2& uv, float3& origin, float3& direction, float& pdf)
{
  const float3 p_sensor = params.camera.origin + uv.x * params.camera.right +
                          uv.y * params.camera.up;
  const float3 p_pinhole =
      params.camera.origin + params.camera.f * params.camera.forward;

  origin = p_pinhole;
  direction = normalize(p_pinhole - p_sensor);
  pdf = 1.0f / dot(direction, params.camera.forward);
}

static __forceinline__ __device__ void fill_surface_info(
    const float3* vertices, const uint3* indices, const float3* normals,
    const float2* texcoords, const float3& ray_origin,
    const float3& ray_direction, float ray_tmax, const float2& barycentric,
    uint prim_idx, SurfaceInfo& info)
{
  info.t = ray_tmax;
  info.x = ray_origin + ray_tmax * ray_direction;
  info.barycentric = barycentric;

  const uint3 idx = indices[prim_idx];
  const float3 v0 = vertices[idx.x];
  const float3 v1 = vertices[idx.y];
  const float3 v2 = vertices[idx.z];
  info.n_g = normalize(cross(v1 - v0, v2 - v0));

  const float3 n0 = normals[idx.x];
  const float3 n1 = normals[idx.y];
  const float3 n2 = normals[idx.z];
  info.n_s = normalize((1.0f - info.barycentric.x - info.barycentric.y) * n0 +
                       info.barycentric.x * n1 + info.barycentric.y * n2);

  const float2 tex0 = texcoords[idx.x];
  const float2 tex1 = texcoords[idx.y];
  const float2 tex2 = texcoords[idx.z];
  info.texcoord = (1.0f - info.barycentric.x - info.barycentric.y) * tex0 +
                  info.barycentric.x * tex1 + info.barycentric.y * tex2;

  // flip normal
  info.is_entering = dot(-ray_direction, info.n_g) > 0;
  info.n_s = info.is_entering ? info.n_s : -info.n_s;
  info.n_g = info.is_entering ? info.n_g : -info.n_g;

  orthonormal_basis(info.n_s, info.tangent, info.bitangent);
}

static __forceinline__ __device__ ShadingParams fill_shading_params(
    const Material& material, const SurfaceInfo& surf_info,
    const hipTextureObject_t* textures, ShadingParams& shading_params)
{
  // base color
  shading_params.base_color =
      material.base_color_texture_id >= 0
          ? make_float3(tex2D<float4>(textures[material.base_color_texture_id],
                                      surf_info.texcoord.x,
                                      surf_info.texcoord.y))
          : material.base_color;

  // specular
  shading_params.specular = material.specular;

  // specular color
  shading_params.specular_color =
      material.specular_color_texture_id >= 0
          ? make_float3(
                tex2D<float4>(textures[material.specular_color_texture_id],
                              surf_info.texcoord.x, surf_info.texcoord.y))
          : material.specular_color;

  // specular roughness
  shading_params.specular_roughness =
      material.specular_roughness_texture_id >= 0
          ? tex2D<float4>(textures[material.specular_roughness_texture_id],
                          surf_info.texcoord.x, surf_info.texcoord.y)
                .x
          : material.specular_roughness;

  // metalness
  shading_params.metalness =
      material.metalness_texture_id >= 0
          ? tex2D<float4>(textures[material.metalness_texture_id],
                          surf_info.texcoord.x, surf_info.texcoord.y)
                .x
          : material.metalness;

  // coat
  shading_params.coat = material.coat;

  // coat roughness
  shading_params.coat_roughness = material.coat_roughness;

  // transmission
  shading_params.transmission = material.transmission;

  // transmission color
  shading_params.transmission_color = material.transmission_color;
}

static __forceinline__ __device__ float3
sample_position_on_light(const float u, const float2& v, const float3* vertices,
                         const uint3* indices, const float3* normals,
                         float3& le, float3& n, float& pdf)
{
  // sample light
  const uint light_idx =
      clamp(static_cast<uint>(params.n_lights * u), 0u, params.n_lights - 1);
  const Light& light = params.lights[light_idx];

  // sample point on the light
  const float2 barycentric = sample_triangle(v);

  const uint3 idx = light.indices;
  const float3 v0 = vertices[idx.x];
  const float3 v1 = vertices[idx.y];
  const float3 v2 = vertices[idx.z];
  const float3 n0 = normals[idx.x];
  const float3 n1 = normals[idx.y];
  const float3 n2 = normals[idx.z];

  const float3 p = (1.0f - barycentric.x - barycentric.y) * v0 +
                   barycentric.x * v1 + barycentric.y * v2;
  n = (1.0f - barycentric.x - barycentric.y) * n0 + barycentric.x * n1 +
      barycentric.y * n2;
  const float area = 0.5f * length(cross(v1 - v0, v2 - v0));

  le = light.le;
  pdf = 1.0f / (params.n_lights * area);

  return p;
}

static __forceinline__ __device__ float3 fetch_ibl(const float3& v)
{
  const float2 thphi = cartesian_to_spherical(v);
  return make_float3(
      tex2D<float4>(params.ibl, thphi.y / (2.0f * M_PIf), thphi.x / M_PIf));
}

static __forceinline__ __device__ float compute_mis_weight(float pdf0,
                                                           float pdf1)
{
  return pdf0 / (pdf0 + pdf1);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();
  const uint image_idx = idx.x + params.width * idx.y;

  // set RNG state
  RadiancePayload payload;
  payload.rng.state = params.rng_states[image_idx].state;
  payload.rng.inc = params.rng_states[image_idx].inc;

  uint n_spp = params.sample_count[image_idx];
  float3 beauty = make_float3(params.render_layer.beauty[image_idx]);
  float3 position = make_float3(params.render_layer.position[image_idx]);
  float3 normal = make_float3(params.render_layer.normal[image_idx]);
  float depth = params.render_layer.depth[image_idx];
  float2 texcoord = make_float2(params.render_layer.texcoord[image_idx]);
  float3 albedo = make_float3(params.render_layer.albedo[image_idx]);

  for (int spp = 0; spp < params.n_samples; ++spp) {
    // generate initial ray from camera
    float2 uv =
        make_float2((2.0f * (idx.x + frandom(payload.rng)) - dim.x) / dim.y,
                    (2.0f * (idx.y + frandom(payload.rng)) - dim.y) / dim.y);
    // flip x
    uv.x = -uv.x;
    float camera_pdf;
    sample_ray_pinhole_camera(uv, payload.origin, payload.direction,
                              camera_pdf);

    // start ray tracing from the camera
    payload.radiance = make_float3(0);
    payload.throughput =
        make_float3(dot(payload.direction, params.camera.forward) / camera_pdf);
    payload.done = false;
    for (int ray_depth = 0; ray_depth < params.max_depth; ++ray_depth) {
      // russian roulette
      const float russian_roulette_prob =
          ray_depth == 0
              ? 1.0f
              : clamp(rgb_to_luminance(payload.throughput), 0.0f, 1.0f);
      if (frandom(payload.rng) >= russian_roulette_prob) { break; }
      payload.throughput /= russian_roulette_prob;

      // trace ray and update payloads
      trace_radiance(params.ias_handle, payload.origin, payload.direction, 0.0f,
                     1e9f, &payload);

      // throughput nan check
      if (isnan(payload.throughput) || isinf(payload.throughput)) { break; }

      if (payload.done) { break; }
    }

    // radiance nan check
    float3 radiance = make_float3(0.0f);
    if (!isnan(payload.radiance) && !isinf(payload.radiance)) {
      radiance = payload.radiance;
    }

    // take streaming average
    const float coef = 1.0f / (n_spp + 1.0f);
    beauty = coef * (n_spp * beauty + radiance);
    position = coef * (n_spp * position + payload.position);
    normal = coef * (n_spp * normal + payload.normal);
    depth = coef * (n_spp * depth + payload.depth);
    texcoord = coef * (n_spp * texcoord + payload.texcoord);
    albedo = coef * (n_spp * albedo + payload.albedo);

    n_spp++;
  }

  // update total number of samples
  params.sample_count[image_idx] = n_spp;

  // save RNG state for next render call
  params.rng_states[image_idx].state = payload.rng.state;
  params.rng_states[image_idx].inc = payload.rng.inc;

  // write results in render layers
  params.render_layer.beauty[image_idx] = make_float4(beauty, 1.0f);
  params.render_layer.position[image_idx] = make_float4(position, 1.0f);
  params.render_layer.normal[image_idx] = make_float4(normal, 1.0f);
  params.render_layer.depth[image_idx] = depth;
  params.render_layer.texcoord[image_idx] = make_float4(texcoord, 0.0f, 1.0f);
  params.render_layer.albedo[image_idx] = make_float4(albedo, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();

  // firsthit light case
  if (payload->firsthit) {
    float3 le;
    if (params.ibl) {
      le = fetch_ibl(payload->direction);
    } else {
      le = params.bg_color;
    }

    payload->radiance += payload->throughput * le;
  }

  payload->done = true;
}

extern "C" __global__ void __miss__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visible = true;
}

extern "C" __global__ void __miss__light()
{
  LightPayload* payload = get_payload_ptr<LightPayload>();

  if (params.ibl) {
    payload->le = fetch_ibl(payload->direction);
  } else {
    payload->le = params.bg_color;
  }
}

extern "C" __global__ void __anyhit__radiance()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.base_color_texture_id],
                      texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id], texcoord.x,
                      texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __anyhit__shadow()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.base_color_texture_id],
                      texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id], texcoord.x,
                      texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __anyhit__light()
{
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  // fill surface info
  const float2 barycentric = optixGetTriangleBarycentrics();

  // calc texcoord
  const uint3 idx = sbt->indices[prim_idx];
  const float2 tex0 = sbt->texcoords[idx.x];
  const float2 tex1 = sbt->texcoords[idx.y];
  const float2 tex2 = sbt->texcoords[idx.z];
  const float2 texcoord = (1.0f - barycentric.x - barycentric.y) * tex0 +
                          barycentric.x * tex1 + barycentric.y * tex2;

  // fetch base color texture
  if (material.base_color_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.base_color_texture_id],
                      texcoord.x, texcoord.y)
            .w;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }

  // fetch alpha texture
  if (material.alpha_texture_id >= 0) {
    const float alpha =
        tex2D<float4>(params.textures[material.alpha_texture_id], texcoord.x,
                      texcoord.y)
            .x;

    // ignore intersection
    if (alpha < 0.5) { optixIgnoreIntersection(); }
  }
}

extern "C" __global__ void __closesthit__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();

  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  const float3 ray_origin = optixGetWorldRayOrigin();
  const float3 ray_direction = optixGetWorldRayDirection();
  const float ray_tmax = optixGetRayTmax();
  const float2 barycentric = optixGetTriangleBarycentrics();

  SurfaceInfo surf_info;
  fill_surface_info(sbt->vertices, sbt->indices, sbt->normals, sbt->texcoords,
                    ray_origin, ray_direction, ray_tmax, barycentric, prim_idx,
                    surf_info);

  ShadingParams shading_params;
  fill_shading_params(material, surf_info, params.textures, shading_params);

  // fill position, normal, depth, albedo
  if (payload->firsthit) {
    payload->position = surf_info.x;
    payload->normal = surf_info.n_s;
    payload->depth = surf_info.t;
    payload->texcoord = surf_info.texcoord;
    payload->albedo = shading_params.base_color;
    payload->firsthit = false;

    // first hit light case
    if (has_emission(material)) {
      payload->radiance += payload->throughput * material.emission_color;
      payload->done = true;
      return;
    }
  }

  // normal mapping
  float3 tangent = surf_info.tangent;
  float3 normal = surf_info.n_s;
  float3 bitangent = surf_info.bitangent;
  if (material.normalmap_texture_id >= 0) {
    float3 value = make_float3(
        tex2D<float4>(params.textures[material.normalmap_texture_id],
                      surf_info.texcoord.x, surf_info.texcoord.y));
    value = normalize(0.5f * (value + 1.0f));
    normal = local_to_world(value, surf_info.tangent, surf_info.n_s,
                            surf_info.bitangent);
    orthonormal_basis(normal, tangent, bitangent);
  }

  const float3 wo = world_to_local(-ray_direction, tangent, normal, bitangent);
  const BSDF bsdf = BSDF(shading_params, surf_info.is_entering);

  // light sampling
  {
    // sky
    if (params.ibl) {
      // TODO: implement IBL importance sampling
      const float3 wi = sample_cosine_weighted_hemisphere(
          make_float2(frandom(payload->rng), frandom(payload->rng)));
      const float3 shadow_ray_origin = surf_info.x + RAY_EPS * surf_info.n_g;
      const float3 shadow_ray_direction =
          local_to_world(wi, tangent, normal, bitangent);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, 1e9f, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 f = bsdf.eval(wo, wi);
        const float pdf = abs_cos_theta(wi) / M_PIf;
        payload->radiance += payload->throughput * f * abs_cos_theta(wi) *
                             fetch_ibl(shadow_ray_direction) / pdf;
      }
    } else {
      const float3 wi = sample_cosine_weighted_hemisphere(
          make_float2(frandom(payload->rng), frandom(payload->rng)));
      const float3 shadow_ray_origin = surf_info.x + RAY_EPS * surf_info.n_g;
      const float3 shadow_ray_direction =
          local_to_world(wi, tangent, normal, bitangent);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, 1e9f, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 f = bsdf.eval(wo, wi);
        const float pdf = abs_cos_theta(wi) / M_PIf;
        payload->radiance +=
            payload->throughput * f * abs_cos_theta(wi) * params.bg_color / pdf;
      }
    }

    // area light
    if (params.n_lights > 0) {
      float3 le, n;
      float pdf_area;
      const float3 p = sample_position_on_light(
          frandom(payload->rng),
          make_float2(frandom(payload->rng), frandom(payload->rng)),
          sbt->vertices, sbt->indices, sbt->normals, le, n, pdf_area);

      const float3 shadow_ray_origin = surf_info.x + RAY_EPS * surf_info.n_g;
      const float3 shadow_ray_direction = normalize(p - shadow_ray_origin);
      const float r = length(p - shadow_ray_origin);

      ShadowPayload shadow_payload;
      trace_shadow(params.ias_handle, shadow_ray_origin, shadow_ray_direction,
                   0.0f, r - RAY_EPS, &shadow_payload);

      if (shadow_payload.visible) {
        const float3 wi =
            world_to_local(shadow_ray_direction, tangent, normal, bitangent);
        const float3 f = bsdf.eval(wo, wi);
        const float pdf =
            r * r / fabs(dot(-shadow_ray_direction, n)) * pdf_area;
        payload->radiance +=
            payload->throughput * f * abs_cos_theta(wi) * le / pdf;
      }
    }
  }

  // BSDF sampling
  {
    const float4 u = make_float4(frandom(payload->rng), frandom(payload->rng),
                                 frandom(payload->rng), frandom(payload->rng));
    const float2 v = make_float2(frandom(payload->rng), frandom(payload->rng));
    float3 f;
    float pdf;
    const float3 wi = bsdf.sample(wo, u, v, f, pdf);

    const float3 light_ray_origin = surf_info.x + RAY_EPS * surf_info.n_g;
    const float3 light_ray_direction =
        local_to_world(wi, tangent, normal, bitangent);

    LightPayload light_payload;
    light_payload.direction = light_ray_direction;
    trace_light(params.ias_handle, light_ray_origin, light_ray_direction, 0.0f,
                1e9f, &light_payload);

    payload->radiance +=
        payload->throughput * f * abs_cos_theta(wi) * light_payload.le / pdf;
  }

  // generate next ray direction
  {
    const float4 u = make_float4(frandom(payload->rng), frandom(payload->rng),
                                 frandom(payload->rng), frandom(payload->rng));
    const float2 v = make_float2(frandom(payload->rng), frandom(payload->rng));
    float3 f;
    float pdf;
    const float3 wi = bsdf.sample(wo, u, v, f, pdf);
    const float3 wi_world = local_to_world(wi, tangent, normal, bitangent);

    // update throughput
    payload->throughput *= f * abs_cos_theta(wi) / pdf;

    // advance ray
    payload->origin = surf_info.x;
    payload->direction = wi_world;

    // adjust ray origin to prevent self-intersection
    const bool is_transmitted = dot(wi_world, surf_info.n_g) < 0;
    if (is_transmitted) {
      payload->origin -= RAY_EPS * surf_info.n_g;
    } else {
      payload->origin += RAY_EPS * surf_info.n_g;
    }
  }
}

extern "C" __global__ void __closesthit__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visible = false;
}

extern "C" __global__ void __closesthit__light()
{
  LightPayload* payload = get_payload_ptr<LightPayload>();

  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());
  const uint prim_idx = optixGetPrimitiveIndex();

  // get material info
  const uint material_id = sbt->material_ids[prim_idx];
  const Material& material = params.materials[material_id];

  if (has_emission(material)) {
    payload->le = material.emission_color;
  } else {
    payload->le = make_float3(0.0f);
  }
}