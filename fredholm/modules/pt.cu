#include "hip/hip_runtime.h"
#include <optix.h>

#include "math.cu"
#include "sampling.cu"
#include "shared.h"
#include "sutil/vec_math.h"

#define RAY_EPS 0.001f

using namespace fredholm;

extern "C" {
__constant__ LaunchParams params;
}

enum class RayType : unsigned int {
  RAY_TYPE_RADIANCE = 0,
  RAY_TYPE_SHADOW = 1,
  RAY_TYPE_COUNT
};

struct RadiancePayload {
  float3 origin;
  float3 direction;

  float3 throughput = make_float3(1);
  float3 radiance = make_float3(0);

  RNGState rng;

  bool done = false;
};

// upper-32bit + lower-32bit -> 64bit
static __forceinline__ __device__ void* unpack_ptr(unsigned int i0,
                                                   unsigned int i1)
{
  const unsigned long long uptr =
      static_cast<unsigned long long>(i0) << 32 | i1;
  void* ptr = reinterpret_cast<void*>(uptr);
  return ptr;
}

// 64bit -> upper-32bit + lower-32bit
static __forceinline__ __device__ void pack_ptr(void* ptr, unsigned int& i0,
                                                unsigned int& i1)
{
  const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
  i0 = uptr >> 32;
  i1 = uptr & 0x00000000ffffffff;
}

// u0, u1 is upper-32bit, lower-32bit of ptr of Payload
template <typename Payload>
static __forceinline__ __device__ Payload* get_payload_ptr()
{
  const unsigned int u0 = optixGetPayload_0();
  const unsigned int u1 = optixGetPayload_1();
  return reinterpret_cast<Payload*>(unpack_ptr(u0, u1));
}

// trace radiance ray
static __forceinline__ __device__ void trace_radiance(
    OptixTraversableHandle& handle, const float3& ray_origin,
    const float3& ray_direction, float tmin, float tmax,
    RadiancePayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE), u0, u1);
}

static __forceinline__ __device__ bool has_emission(const Material& material)
{
  return (material.emission_color.x > 0 || material.emission_color.y > 0 ||
          material.emission_color.z > 0);
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    const float2& uv, float3& origin, float3& direction)
{
  const float3 p_sensor = params.camera.origin + uv.x * params.camera.right +
                          uv.y * params.camera.up;
  const float3 p_pinhole =
      params.camera.origin + params.camera.f * params.camera.forward;

  origin = p_sensor;
  direction = normalize(p_pinhole - p_sensor);
}

static __forceinline__ __device__ void fill_surface_info(
    const float3* vertices, const uint3* indices, const float3* normals,
    const float2* texcoords, const float3& ray_origin,
    const float3& ray_direction, float ray_tmax, const float2& barycentric,
    uint prim_idx, SurfaceInfo& info)
{
  info.x = ray_origin + ray_tmax * ray_direction;
  info.barycentric = barycentric;

  const uint3 idx = indices[prim_idx];
  const float3 v0 = vertices[idx.x];
  const float3 v1 = vertices[idx.y];
  const float3 v2 = vertices[idx.z];
  info.n_g = normalize(cross(v1 - v0, v2 - v0));

  const float3 n0 = normals[idx.x];
  const float3 n1 = normals[idx.y];
  const float3 n2 = normals[idx.z];
  info.n_s = (1.0f - info.barycentric.x - info.barycentric.y) * n0 +
             info.barycentric.x * n1 + info.barycentric.y * n2;

  const float2 tex0 = texcoords[idx.x];
  const float2 tex1 = texcoords[idx.y];
  const float2 tex2 = texcoords[idx.z];
  info.texcoord = (1.0f - info.barycentric.x - info.barycentric.y) * tex0 +
                  info.barycentric.x * tex1 + info.barycentric.y * tex2;

  orthonormal_basis(info.n_s, info.tangent, info.bitangent);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();
  const uint image_idx = idx.x + params.width * idx.y;

  // set RNG state
  RadiancePayload payload;
  payload.rng.state = params.rng_states[image_idx].state;
  payload.rng.inc = params.rng_states[image_idx].inc;

  for (int spp = 0; spp < params.n_samples; ++spp) {
    // generate initial ray from camera
    const float2 uv =
        make_float2((2.0f * (idx.x + frandom(payload.rng)) - dim.x) / dim.x,
                    (2.0f * (idx.y + frandom(payload.rng)) - dim.y) / dim.y);
    sample_ray_pinhole_camera(uv, payload.origin, payload.direction);

    // start ray tracing from the camera
    payload.radiance = make_float3(0);
    payload.throughput = make_float3(1);
    payload.done = false;
    for (int depth = 0; depth < params.max_depth; ++depth) {
      trace_radiance(params.gas_handle, payload.origin, payload.direction, 0.0f,
                     1e9f, &payload);

      if (payload.done) { break; }
    }

    // accumulate contribution
    params.accumulation[image_idx] += make_float4(payload.radiance, 1.0f);
    params.sample_count[image_idx] += 1;
  }

  // save RNG state for next sampling
  params.rng_states[image_idx].state = payload.rng.state;
  params.rng_states[image_idx].inc = payload.rng.inc;

  // take average
  float3 radiance = make_float3(params.accumulation[image_idx]);
  radiance /= params.sample_count[image_idx];

  // gamma correction
  radiance.x = pow(radiance.x, 1.0f / 2.2f);
  radiance.y = pow(radiance.y, 1.0f / 2.2f);
  radiance.z = pow(radiance.z, 1.0f / 2.2f);

  // write radiance to frame buffer
  params.framebuffer[image_idx] = make_float4(radiance, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
  const MissSbtRecordData* sbt =
      reinterpret_cast<MissSbtRecordData*>(optixGetSbtDataPointer());

  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  payload->radiance += payload->throughput * sbt->bg_color;
  payload->done = true;
}

extern "C" __global__ void __miss__shadow() {}

extern "C" __global__ void __closesthit__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());

  // Le
  if (has_emission(sbt->material)) {
    payload->radiance += payload->throughput * sbt->material.emission_color;
    payload->done = true;
    return;
  }

  const float3 ray_origin = optixGetWorldRayOrigin();
  const float3 ray_direction = optixGetWorldRayDirection();
  const float ray_tmax = optixGetRayTmax();
  const float2 barycentric = optixGetTriangleBarycentrics();
  const uint prim_idx = optixGetPrimitiveIndex();

  SurfaceInfo surf_info;
  fill_surface_info(sbt->vertices, sbt->indices, sbt->normals, sbt->texcoords,
                    ray_origin, ray_direction, ray_tmax, barycentric, prim_idx,
                    surf_info);

  // sample next ray direction
  const float3 wi = sample_cosine_weighted_hemisphere(frandom(payload->rng),
                                                      frandom(payload->rng));
  const float3 wi_world =
      local_to_world(wi, surf_info.tangent, surf_info.n_s, surf_info.bitangent);

  // update payload
  payload->throughput *= sbt->material.base_color;

  // advance ray
  payload->origin = surf_info.x + RAY_EPS * surf_info.n_s;
  payload->direction = wi_world;
}

extern "C" __global__ void __closesthit__shadow() {}