#include "hip/hip_runtime.h"
#include "sutil/vec_math.h"

__forceinline__ __device__ float cos_theta(const float3& w) { return w.y; }

__forceinline__ __device__ float cos2_theta(const float3& w)
{
  return w.y * w.y;
}

__forceinline__ __device__ float abs_cos_theta(const float3& w)
{
  return fabs(w.y * w.y);
}

__forceinline__ __device__ float sin_theta(const float3& w)
{
  return sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_theta(const float3& w)
{
  return fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_theta(const float3& w)
{
  return fabs(sin_theta(w));
}

__forceinline__ __device__ float tan_theta(const float3& w)
{
  return sin_theta(w) / cos_theta(w);
}

__forceinline__ __device__ float tan2_theta(const float3& w)
{
  return 1.0f / (w.y * w.y) - 1.0f;
}

__forceinline__ __device__ float abs_tan_theta(const float3& w)
{
  return fabs(tan_theta(w));
}

__forceinline__ __device__ float sin_phi(const float3& w)
{
  return w.z / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_phi(const float3& w)
{
  return w.z * w.z / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_phi(const float3& w)
{
  return fabs(sin_phi(w));
}

__forceinline__ __device__ float cos_phi(const float3& w)
{
  return w.x / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float cos2_phi(const float3& w)
{
  return w.x * w.x / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_cos_phi(const float3& w)
{
  return fabs(cos_phi(w));
}

class MicrofacetReflection
{
 public:
  __device__ MicrofacetReflection() {}

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    const float3 f = make_float3(1.0f);
    const float d = D(wh);
    const float g = G2(wo, wi);
    return 0.25f * (f * d * g) / (abs_cos_theta(wo) * abs_cos_theta(wi));
  }

  __device__ float3 sample(const float3& wo, float3& wi, float& pdf) const {}

  __device__ float pdf(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    return 0.25f * D_visible(wo, wh) / abs_cos_theta(wo);
  }

 private:
  __device__ float D(const float3& wh) const
  {
    const float t = wh.x * wh.x / (alpha.x * alpha.x) +
                    wh.z * wh.z / (alpha.y * alpha.y) + wh.y * wh.y;
    return 1.0f / (M_PI * alpha.x * alpha.y * t * t);
  }

  __device__ float D_visible(const float3& w, const float3& wh) const
  {
    return G1(w) * fabs(dot(w, wh)) * D(wh) / abs_cos_theta(w);
  }

  __device__ float lambda(const float3& w) const
  {
    const float a0 = sqrtf(cos2_phi(w) * alpha.x * alpha.x +
                           sin2_phi(w) * alpha.y * alpha.y);
    const float a = 1.0f / (a0 * tan_theta(w));
    return 0.5f * (-1.0f + sqrtf(1.0f + 1.0f / (a * a)));
  }

  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  // https://jcgt.org/published/0007/04/01/
  __device__ float3 sample_vndf(const float3& wo, const float2& u) const
  {
    const float3 Vh =
        normalize(make_float3(alpha.x * wo.x, wo.y, alpha.y * wo.z));

    const float lensq = Vh.x * Vh.x + Vh.z * Vh.z;
    const float3 T1 = lensq > 0 ? make_float3(Vh.z, 0, -Vh.x) / sqrtf(lensq)
                                : make_float3(0, 0, 1);
    const float3 T2 = cross(Vh, T1);

    const float r = sqrtf(u.x);
    const float phi = 2.0f * M_PI * u.y;
    const float t1 = r * cosf(phi);
    float t2 = r * sinf(phi);
    const float s = 0.5f * (1.0f + Vh.y);
    t2 = (1.0f - s) * sqrtf(fmax(1.0f - t1 * t1, 0.0f)) + s * t2;
    const float3 Nh =
        t1 * T1 + t2 * T2 + sqrtf(fmax(1.0f - t1 * t1 - t2 * t2, 0.0f)) * Vh;
    const float3 Ne = normalize(
        make_float3(alpha.x * Nh.x, fmax(0.0f, Nh.y), alpha.y * Nh.z));

    return Ne;
  }

  float2 alpha;
};