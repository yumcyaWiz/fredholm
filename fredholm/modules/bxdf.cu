#include "hip/hip_runtime.h"
#pragma once

#include <cmath>

#include "math.cu"
#include "sampling.cu"
#include "sutil/vec_math.h"

__forceinline__ __device__ float cos_theta(const float3& w) { return w.y; }

__forceinline__ __device__ float cos2_theta(const float3& w)
{
  return w.y * w.y;
}

__forceinline__ __device__ float abs_cos_theta(const float3& w)
{
  return fabs(w.y);
}

__forceinline__ __device__ float sin_theta(const float3& w)
{
  return sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_theta(const float3& w)
{
  return fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_theta(const float3& w)
{
  return fabs(sin_theta(w));
}

__forceinline__ __device__ float tan_theta(const float3& w)
{
  return sin_theta(w) / cos_theta(w);
}

__forceinline__ __device__ float tan2_theta(const float3& w)
{
  return 1.0f / (w.y * w.y) - 1.0f;
}

__forceinline__ __device__ float abs_tan_theta(const float3& w)
{
  return fabs(tan_theta(w));
}

__forceinline__ __device__ float sin_phi(const float3& w)
{
  return w.z / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_phi(const float3& w)
{
  return w.z * w.z / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_phi(const float3& w)
{
  return fabs(sin_phi(w));
}

__forceinline__ __device__ float cos_phi(const float3& w)
{
  return w.x / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float cos2_phi(const float3& w)
{
  return w.x * w.x / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_cos_phi(const float3& w)
{
  return fabs(cos_phi(w));
}

__forceinline__ __device__ float3 reflect(const float3& w, const float3& n)
{
  return normalize(-w + 2.0f * dot(w, n) * n);
}

__forceinline__ __device__ bool refract(const float3& w, const float3& n,
                                        float ior_i, float ior_t, float3& wt)
{
  const float3 th = -ior_i / ior_t * (w - dot(w, n) * n);
  if (dot(th, th) > 1.0f) return false;
  const float3 tp = -sqrtf(fmax(1.0f - dot(th, th), 0.0f)) * n;
  wt = th + tp;
  return true;
}

__forceinline__ __device__ float2 roughness_to_alpha(float roughness,
                                                     float anisotropy)
{
  // Revisiting Physically Based Shading at Imageworks p.24
  float2 alpha;
  alpha.x = roughness * roughness * (1.0f + anisotropy);
  alpha.y = roughness * roughness * (1.0f - anisotropy);
  return alpha;
}

// https://jcgt.org/published/0003/04/03/
__forceinline__ __device__ float3 artist_friendly_metallic_fresnel(
    const float3& reflectivity, const float3& edge_tint, float3& n, float3& k)
{
  const float3 r_sqrt = sqrt(reflectivity);
  n = edge_tint * (1.0f - reflectivity) / (1.0f + reflectivity) +
      (1.0f - edge_tint) * (1.0f + r_sqrt) / (1.0f - r_sqrt);
  const float3 t1 = n + 1.0f;
  const float3 t2 = n - 1.0f;
  k = sqrt((reflectivity * (t1 * t1) - t2 * t2) / (1.0f - reflectivity));
}

class Lambert
{
 public:
  __device__ Lambert() {}
  __device__ Lambert(const float3& albedo) : m_albedo(albedo) {}

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    return m_albedo / M_PIf;
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    const float3 wi = sample_cosine_weighted_hemisphere(u);

    f = eval(wo, wi);
    pdf = abs_cos_theta(wi) / M_PIf;

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    return abs_cos_theta(wi) / M_PIf;
  }

 private:
  float3 m_albedo;
};

class OrenNayer
{
 public:
  __device__ OrenNayer() {}
  __device__ OrenNayer(const float3& albedo, float roughness)
      : m_albedo(albedo), m_roughness(roughness)
  {
    const float sigma2 = roughness * roughness;
    m_A = 1.0f - (sigma2 / (2.0f * (sigma2 + 0.33f)));
    m_B = 0.45f * sigma2 / (sigma2 + 0.09f);
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float s_theta_o = sin_theta(wo);
    const float s_theta_i = sin_theta(wi);

    float c_max = 0.0f;
    if (s_theta_i > 1e-4f && s_theta_o > 1e-4f) {
      const float s_phi_o = sin_phi(wo), c_phi_o = cos_phi(wo);
      const float s_phi_i = sin_phi(wi), c_phi_i = cos_phi(wi);
      const float c = c_phi_i * c_phi_o + s_phi_i * s_phi_o;
      c_max = fmax(c, 0.0f);
    }

    const bool b = abs_cos_theta(wi) > abs_cos_theta(wo);
    const float s_alpha = b ? s_theta_o : s_theta_i;
    const float t_beta =
        b ? s_theta_i / abs_cos_theta(wi) : s_theta_o / abs_cos_theta(wo);

    return m_albedo * (m_A + m_B * c_max * s_alpha * t_beta) / M_PIf;
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    const float3 wi = sample_cosine_weighted_hemisphere(u);

    f = eval(wo, wi);
    pdf = abs_cos_theta(wi) / M_PIf;

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    return abs_cos_theta(wi) / M_PIf;
  }

 private:
  float3 m_albedo;
  float m_roughness;
  float m_A;
  float m_B;
};

// The diffuse transmission is modeled via a Oren-Nayar microfacet BRDF flipped
// about the shading normal to make it a BTDF
class DiffuseTransmission
{
 public:
  __device__ DiffuseTransmission() {}
  __device__ DiffuseTransmission(const float3& albedo, float roughness)
      : m_albedo(albedo), m_roughness(roughness)
  {
    const float sigma2 = roughness * roughness;
    m_A = 1.0f - (sigma2 / (2.0f * (sigma2 + 0.33f)));
    m_B = 0.45f * sigma2 / (sigma2 + 0.09f);
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float s_theta_o = sin_theta(wo);
    const float s_theta_i = sin_theta(wi);

    float c_max = 0.0f;
    if (s_theta_i > 1e-4f && s_theta_o > 1e-4f) {
      const float s_phi_o = sin_phi(wo), c_phi_o = cos_phi(wo);
      const float s_phi_i = sin_phi(wi), c_phi_i = cos_phi(wi);
      const float c = c_phi_i * c_phi_o + s_phi_i * s_phi_o;
      c_max = fmax(c, 0.0f);
    }

    const bool b = abs_cos_theta(wi) > abs_cos_theta(wo);
    const float s_alpha = b ? s_theta_o : s_theta_i;
    const float t_beta =
        b ? s_theta_i / abs_cos_theta(wi) : s_theta_o / abs_cos_theta(wo);

    return m_albedo * (m_A + m_B * c_max * s_alpha * t_beta) / M_PIf;
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    float3 wi = sample_cosine_weighted_hemisphere(u);
    // flip direction
    wi = -wi;

    f = eval(wo, wi);
    pdf = abs_cos_theta(wi) / M_PIf;

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    return abs_cos_theta(wi) / M_PIf;
  }

 private:
  float3 m_albedo;
  float m_roughness;
  float m_A;
  float m_B;
};

struct FresnelSchlick {
  __device__ FresnelSchlick() {}
  __device__ FresnelSchlick(float n)
  {
    const float t = (1.0f - n) / (1.0f + n);
    m_F0 = t * t;
  }

  __device__ float eval(float cos) const
  {
    const float t = fmax(1.0f - cos, 0.0f);
    return m_F0 + fmax(1.0f - m_F0, 0.0f) * t * t * t * t * t;
  }

  float m_F0;
};

struct FresnelDielectric {
  __device__ FresnelDielectric() {}
  __device__ FresnelDielectric(float n) : m_n(n) {}

  __device__ float eval(float cos) const
  {
    const float temp = m_n * m_n + cos * cos - 1.0f;
    if (temp < 0.0f) { return 1.0f; }

    const float g = sqrtf(temp);
    const float t0 = (g - cos) / (g + cos);
    const float t1 = ((g + cos) * cos - 1.0f) / ((g - cos) * cos + 1.0f);
    return 0.5f * t0 * t0 * (1.0f + t1 * t1);
  }

  float m_n;
};

struct FresnelConductor {
  __device__ FresnelConductor() {}
  __device__ FresnelConductor(const float3& n, const float3& k) : m_n(n), m_k(k)
  {
  }

  __device__ float3 eval(float cos) const
  {
    const float c2 = cos * cos;
    const float3 two_eta_cos = 2.0f * m_n * cos;

    const float3 t0 = m_n * m_n + m_k * m_k;
    const float3 t1 = t0 * c2;
    const float3 Rs = (t0 - two_eta_cos + c2) / (t0 + two_eta_cos + c2);
    const float3 Rp = (t1 - two_eta_cos + 1.0f) / (t1 + two_eta_cos + 1.0f);

    return 0.5f * (Rp + Rs);
  }

  float3 m_n;
  float3 m_k;
};

class MicrofacetReflectionDielectric
{
 public:
  __device__ MicrofacetReflectionDielectric() {}
  __device__ MicrofacetReflectionDielectric(float ior, float roughness,
                                            float anisotropy)
      : m_fresnel(ior)
  {
    m_alpha = roughness_to_alpha(roughness, anisotropy);
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    const float f = m_fresnel.eval(fabs(dot(wo, wh)));
    const float d = D(wh);
    const float g = G2(wo, wi);
    return make_float3(0.25f * (f * d * g) /
                       (abs_cos_theta(wo) * abs_cos_theta(wi)));
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    // sample half-vector
    const float3 wh = sample_vndf(wo, m_alpha, u);

    // compute incident direction
    const float3 wi = reflect(wo, wh);

    // evaluate BxDF and pdf
    f = eval(wo, wi);
    pdf = eval_pdf(wo, wi);

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    return 0.25f * D_visible(wo, wh) / fabs(dot(wo, wh));
  }

 private:
  __device__ float D(const float3& wh) const
  {
    const float t = wh.x * wh.x / (m_alpha.x * m_alpha.x) +
                    wh.z * wh.z / (m_alpha.y * m_alpha.y) + wh.y * wh.y;
    return 1.0f / (M_PI * m_alpha.x * m_alpha.y * t * t);
  }

  __device__ float D_visible(const float3& w, const float3& wh) const
  {
    return G1(w) * fabs(dot(w, wh)) * D(wh) / abs_cos_theta(w);
  }

  __device__ float lambda(const float3& w) const
  {
    const float a2 = (cos2_phi(w) * m_alpha.x * m_alpha.x +
                      sin2_phi(w) * m_alpha.y * m_alpha.y);
    const float t = 1.0f / (a2 * tan2_theta(w));
    return 0.5f * (-1.0f + sqrtf(1.0f + 1.0f / t));
  }

  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  FresnelDielectric m_fresnel;
  float2 m_alpha;
};

class MicrofacetReflectionConductor
{
 public:
  __device__ MicrofacetReflectionConductor() {}
  __device__ MicrofacetReflectionConductor(const float3& n, const float3& k,
                                           float roughness, float anisotropy)
      : m_fresnel(n, k)
  {
    m_alpha = roughness_to_alpha(roughness, anisotropy);
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    const float3 f = m_fresnel.eval(fabs(dot(wo, wh)));
    const float d = D(wh);
    const float g = G2(wo, wi);
    return 0.25f * (f * d * g) / (abs_cos_theta(wo) * abs_cos_theta(wi));
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    // sample half-vector
    const float3 wh = sample_vndf(wo, m_alpha, u);

    // compute incident direction
    const float3 wi = reflect(wo, wh);

    // evaluate BxDF and pdf
    f = eval(wo, wi);
    pdf = eval_pdf(wo, wi);

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    return 0.25f * D_visible(wo, wh) / fabs(dot(wo, wh));
  }

 private:
  __device__ float D(const float3& wh) const
  {
    const float t = wh.x * wh.x / (m_alpha.x * m_alpha.x) +
                    wh.z * wh.z / (m_alpha.y * m_alpha.y) + wh.y * wh.y;
    return 1.0f / (M_PI * m_alpha.x * m_alpha.y * t * t);
  }

  __device__ float D_visible(const float3& w, const float3& wh) const
  {
    return G1(w) * fabs(dot(w, wh)) * D(wh) / abs_cos_theta(w);
  }

  __device__ float lambda(const float3& w) const
  {
    const float a2 = (cos2_phi(w) * m_alpha.x * m_alpha.x +
                      sin2_phi(w) * m_alpha.y * m_alpha.y);
    const float t = 1.0f / (a2 * tan2_theta(w));
    return 0.5f * (-1.0f + sqrtf(1.0f + 1.0f / t));
  }

  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  FresnelConductor m_fresnel;
  float2 m_alpha;
};

// Walter, Bruce, et al. "Microfacet Models for Refraction through Rough
// Surfaces." Rendering techniques 2007 (2007): 18th.
class MicrofacetTransmission
{
 public:
  __device__ MicrofacetTransmission() {}
  __device__ MicrofacetTransmission(float ior_i, float ior_t, float roughness,
                                    float anisotropy)
      : m_ior_i(ior_i), m_ior_t(ior_t), m_fresnel(ior_t / ior_i)
  {
    m_alpha = roughness_to_alpha(roughness, anisotropy);
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = compute_half_vector(wo, wi);
    const float f = m_fresnel.eval(fabs(dot(wo, wh)));
    const float d = D(wh);
    const float g = G2(wo, wi);
    const float wo_dot_wh = dot(wo, wh);
    const float wi_dot_wh = dot(wi, wh);
    const float t = m_ior_i * wo_dot_wh + m_ior_t * wi_dot_wh;
    return make_float3(fabs(wo_dot_wh) * fabs(wi_dot_wh) * m_ior_t * m_ior_t *
                       fmax(1.0f - f, 0.0f) * g * d /
                       (abs_cos_theta(wo) * abs_cos_theta(wi) * t * t));
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    // sample half-vector
    const float3 wh = sample_vndf(wo, m_alpha, u);

    // compute incident direction
    float3 wi;
    if (!refract(wo, wh, m_ior_i, m_ior_t, wi)) {
      // total internal reflection
      f = make_float3(0.0f);
      pdf = 1;
      return wi;
    }

    // evaluate BxDF and pdf
    f = eval(wo, wi);
    pdf = eval_pdf(wo, wi);

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    const float3 wh = compute_half_vector(wo, wi);
    const float wi_dot_wh = dot(wi, wh);
    const float t = m_ior_i * dot(wo, wh) + m_ior_t * wi_dot_wh;
    return D_visible(wo, wh) * m_ior_t * m_ior_t * fabs(wi_dot_wh) / (t * t);
  }

 private:
  __device__ float3 compute_half_vector(const float3& wo,
                                        const float3& wi) const
  {
    float3 wh = normalize(-(m_ior_i * wo + m_ior_t * wi));
    if (wh.y < 0.0f) wh = -wh;
    return wh;
  }

  __device__ float D(const float3& wh) const
  {
    const float t = wh.x * wh.x / (m_alpha.x * m_alpha.x) +
                    wh.z * wh.z / (m_alpha.y * m_alpha.y) + wh.y * wh.y;
    return 1.0f / (M_PI * m_alpha.x * m_alpha.y * t * t);
  }

  __device__ float D_visible(const float3& w, const float3& wh) const
  {
    return G1(w) * fabs(dot(w, wh)) * D(wh) / abs_cos_theta(w);
  }

  __device__ float lambda(const float3& w) const
  {
    const float a2 = (cos2_phi(w) * m_alpha.x * m_alpha.x +
                      sin2_phi(w) * m_alpha.y * m_alpha.y);
    const float t = 1.0f / (a2 * tan2_theta(w));
    return 0.5f * (-1.0f + sqrtf(1.0f + 1.0f / t));
  }

  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  float m_ior_i;
  float m_ior_t;
  FresnelDielectric m_fresnel;
  float2 m_alpha;
};

// Production Friendly Microfacet Sheen BRDF
class MicrofacetSheen
{
 public:
  __device__ MicrofacetSheen() {}
  __device__ MicrofacetSheen(float roughness, float eta)
      : m_fresnel(eta), m_roughness(roughness)
  {
  }

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    const float f = m_fresnel.eval(fabs(dot(wo, wh)));
    const float d = D(wh);
    const float g = G2(wo, wi);
    return make_float3(0.25f * (f * d * g) /
                       (abs_cos_theta(wo) * abs_cos_theta(wi)));
  }

  __device__ float3 sample(const float3& wo, const float2& u, float3& f,
                           float& pdf) const
  {
    // sample half-vector
    const float3 wh = sample_cosine_weighted_hemisphere(u);

    // compute incident direction
    const float3 wi = reflect(wo, wh);

    // evaluate BxDF and pdf
    f = eval(wo, wi);
    pdf = eval_pdf(wo, wi);

    return wi;
  }

  __device__ float eval_pdf(const float3& wo, const float3& wi) const
  {
    return abs_cos_theta(wi) / M_PIf;
  }

 private:
  __device__ float D(const float3& wh) const
  {
    const float s = abs_sin_theta(wh);
    return (2.0f + 1.0f / m_roughness) * powf(s, 1.0f / m_roughness) /
           (2.0f * M_PIf);
  }

  __device__ float lambda(const float3& w) const
  {
    const float cos = abs_cos_theta(w);
    if (cos < 0.5f) {
      return expf(L(cos));
    } else {
      return expf(2.0f * L(0.5f) - L(1.0f - cos));
    }
  }
  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  __device__ float L(float x) const
  {
    const auto interpolate = [](float roughness, float p0, float p1) {
      const float t = (1.0f - roughness);
      const float t2 = t * t;
      return t2 * p0 + (1.0f - t2) * p1;
    };

    const float a = interpolate(m_roughness, 25.3245, 21.5473);
    const float b = interpolate(m_roughness, 3.32435, 3.82987);
    const float c = interpolate(m_roughness, 0.16801, 0.19823);
    const float d = interpolate(m_roughness, -1.27393, -1.97760);
    const float e = interpolate(m_roughness, -4.85967, -4.32054);

    return a / (1.0f + b * powf(x, c)) + d * x + e;
  }

  float m_roughness;
  FresnelDielectric m_fresnel;
};