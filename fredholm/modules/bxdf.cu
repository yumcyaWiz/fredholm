
#include <hip/hip_runtime.h>
__forceinline__ __device__ float cos_theta(const float3& w) { return w.y; }

__forceinline__ __device__ float cos2_theta(const float3& w)
{
  return w.y * w.y;
}

__forceinline__ __device__ float abs_cos_theta(const float3& w)
{
  return fabs(w.y * w.y);
}

__forceinline__ __device__ float sin_theta(const float3& w)
{
  return sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_theta(const float3& w)
{
  return fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_theta(const float3& w)
{
  return fabs(sin_theta(w));
}

__forceinline__ __device__ float tan_theta(const float3& w)
{
  return sin_theta(w) / cos_theta(w);
}

__forceinline__ __device__ float tan2_theta(const float3& w)
{
  return 1.0f / (w.y * w.y) - 1.0f;
}

__forceinline__ __device__ float abs_tan_theta(const float3& w)
{
  return fabs(tan_theta(w));
}

__forceinline__ __device__ float sin_phi(const float3& w)
{
  return w.z / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_phi(const float3& w)
{
  return w.z * w.z / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_phi(const float3& w)
{
  return fabs(sin_phi(w));
}

__forceinline__ __device__ float cos_phi(const float3& w)
{
  return w.x / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float cos2_phi(const float3& w)
{
  return w.x * w.x / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_cos_phi(const float3& w)
{
  return fabs(cos_phi(w));
}

struct MicrofacetReflection {
  __device__ MicrofacetReflection() {}

  __device__ float D(const float3& w)
  {
    const float t = w.x * w.x / (alpha.x * alpha.x) +
                    w.z * w.z / (alpha.y * alpha.y) + w.y * w.y;
    return 1.0f / (M_PI * alpha.x * alpha.y * t * t);
  }

  __device__ float G(const float3& w) {}

  __device__ float lambda(const float3& w) {}

  float2 alpha;
};