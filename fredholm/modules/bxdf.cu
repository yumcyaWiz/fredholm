#include "hip/hip_runtime.h"
#include "sutil/vec_math.h"

__forceinline__ __device__ float cos_theta(const float3& w) { return w.y; }

__forceinline__ __device__ float cos2_theta(const float3& w)
{
  return w.y * w.y;
}

__forceinline__ __device__ float abs_cos_theta(const float3& w)
{
  return fabs(w.y * w.y);
}

__forceinline__ __device__ float sin_theta(const float3& w)
{
  return sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_theta(const float3& w)
{
  return fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_theta(const float3& w)
{
  return fabs(sin_theta(w));
}

__forceinline__ __device__ float tan_theta(const float3& w)
{
  return sin_theta(w) / cos_theta(w);
}

__forceinline__ __device__ float tan2_theta(const float3& w)
{
  return 1.0f / (w.y * w.y) - 1.0f;
}

__forceinline__ __device__ float abs_tan_theta(const float3& w)
{
  return fabs(tan_theta(w));
}

__forceinline__ __device__ float sin_phi(const float3& w)
{
  return w.z / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float sin2_phi(const float3& w)
{
  return w.z * w.z / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_sin_phi(const float3& w)
{
  return fabs(sin_phi(w));
}

__forceinline__ __device__ float cos_phi(const float3& w)
{
  return w.x / sqrtf(fmax(1.0f - w.y * w.y, 0.0f));
}

__forceinline__ __device__ float cos2_phi(const float3& w)
{
  return w.x * w.x / fmax(1.0f - w.y * w.y, 0.0f);
}

__forceinline__ __device__ float abs_cos_phi(const float3& w)
{
  return fabs(cos_phi(w));
}

struct MicrofacetReflection {
  __device__ MicrofacetReflection() {}

  __device__ float3 eval(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    const float3 f = make_float3(1.0f);
    const float d = D(wh);
    const float g = G2(wo, wi);
    return 0.25f * (f * d * g) / (abs_cos_theta(wo) * abs_cos_theta(wi));
  }

  __device__ float pdf(const float3& wo, const float3& wi) const
  {
    const float3 wh = normalize(wo + wi);
    return 0.25f * D_visible(wo, wh) / abs_cos_theta(wo);
  }

  __device__ float D(const float3& wh) const
  {
    const float t = wh.x * wh.x / (alpha.x * alpha.x) +
                    wh.z * wh.z / (alpha.y * alpha.y) + wh.y * wh.y;
    return 1.0f / (M_PI * alpha.x * alpha.y * t * t);
  }

  __device__ float D_visible(const float3& w, const float3& wh) const
  {
    return G1(w) * fabs(dot(w, wh)) * D(wh) / abs_cos_theta(w);
  }

  __device__ float lambda(const float3& w) const
  {
    const float a0 = sqrtf(cos2_phi(w) * alpha.x * alpha.x +
                           sin2_phi(w) * alpha.y * alpha.y);
    const float a = 1.0f / (a0 * tan_theta(w));
    return 0.5f * (-1.0f + sqrtf(1.0f + 1.0f / (a * a)));
  }

  __device__ float G1(const float3& w) const
  {
    return 1.0f / (1.0f + lambda(w));
  }

  __device__ float G2(const float3& wo, const float3& wi) const
  {
    return 1.0f / (1.0f + lambda(wo) + lambda(wi));
  }

  float2 alpha;
};