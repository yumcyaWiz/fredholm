#include "hip/hip_runtime.h"
#include "shared.h"

namespace fredholm
{

// *Really* minimal PCG32 code / (c) 2014 M.E. O'Neill / pcg-random.org
// Licensed under Apache License 2.0 (NO WARRANTY, etc. see website)
static __forceinline__ __device__ uint pcg32_random_r(RNGState* rng)
{
  unsigned long long oldstate = rng->state;
  // Advance internal state
  rng->state = oldstate * 6364136223846793005ULL + (rng->inc | 1);
  // Calculate output function (XSH RR), uses old state for max ILP
  uint xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
  uint rot = oldstate >> 59u;
  return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

static __forceinline__ __device__ float frandom(RNGState& rng)
{
  return pcg32_random_r(&rng) / static_cast<float>(0xffffffffu);
}

static __forceinline__ __device__ float3
sample_cosine_weighted_hemisphere(const float u1, const float u2)
{
  // Uniformly sample disk.
  const float r = sqrtf(u1);
  const float phi = 2.0f * M_PIf * u2;

  float3 p;
  p.x = r * cosf(phi);
  p.z = r * sinf(phi);
  // Project up to hemisphere.
  p.y = sqrtf(fmaxf(0.0f, 1.0f - p.x * p.x - p.z * p.z));

  return p;
}

}  // namespace fredholm