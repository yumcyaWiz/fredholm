#include "hip/hip_runtime.h"
#include <optix.h>

#include "math.cu"
#include "sampling.cu"
#include "shared.h"
#include "sutil/vec_math.h"

#define RAY_EPS 0.001f

extern "C" {
__constant__ LaunchParams params;
}

enum class RayType : unsigned int {
  RAY_TYPE_RADIANCE = 0,
  RAY_TYPE_SHADOW = 1,
  RAY_TYPE_COUNT
};

struct RadiancePayload {
  float3 origin;
  float3 direction;
  float3 throughput = make_float3(1);
  RNGState rng;
  bool done = false;
};

struct ShadowPayload {
  float visibility = 0.0f;
};

// upper-32bit + lower-32bit -> 64bit
static __forceinline__ __device__ void* unpack_ptr(unsigned int i0,
                                                   unsigned int i1)
{
  const unsigned long long uptr =
      static_cast<unsigned long long>(i0) << 32 | i1;
  void* ptr = reinterpret_cast<void*>(uptr);
  return ptr;
}

// 64bit -> upper-32bit + lower-32bit
static __forceinline__ __device__ void pack_ptr(void* ptr, unsigned int& i0,
                                                unsigned int& i1)
{
  const unsigned long long uptr = reinterpret_cast<unsigned long long>(ptr);
  i0 = uptr >> 32;
  i1 = uptr & 0x00000000ffffffff;
}

// u0, u1 is upper-32bit, lower-32bit of ptr of Payload
template <typename Payload>
static __forceinline__ __device__ Payload* get_payload_ptr()
{
  const unsigned int u0 = optixGetPayload_0();
  const unsigned int u1 = optixGetPayload_1();
  return reinterpret_cast<Payload*>(unpack_ptr(u0, u1));
}

// trace radiance ray
static __forceinline__ __device__ void trace_radiance(
    OptixTraversableHandle handle, float3 ray_origin, float3 ray_direction,
    float tmin, float tmax, RadiancePayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_NONE,
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_RADIANCE), u0, u1);
}

// trace shadow ray
static __forceinline__ __device__ void trace_shadow(
    OptixTraversableHandle handle, float3 ray_origin, float3 ray_direction,
    float tmin, float tmax, ShadowPayload* payload_ptr)
{
  unsigned int u0, u1;
  pack_ptr(payload_ptr, u0, u1);
  optixTrace(handle, ray_origin, ray_direction, tmin, tmax, 0.0f,
             OptixVisibilityMask(1), OPTIX_RAY_FLAG_TERMINATE_ON_FIRST_HIT,
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW),
             static_cast<unsigned int>(RayType::RAY_TYPE_COUNT),
             static_cast<unsigned int>(RayType::RAY_TYPE_SHADOW), u0, u1);
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    float2 uv, float3& origin, float3& direction)
{
  const float3 p_sensor =
      params.cam_origin + uv.x * params.cam_right + uv.y * params.cam_up;
  const float3 p_pinhole = params.cam_origin + params.cam_forward;

  origin = params.cam_origin;
  direction = normalize(p_pinhole - p_sensor);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();

  float3 radiance = make_float3(0);

  // warm up rng
  // TODO: use some hash function to set more nice seed
  RadiancePayload payload;
  payload.rng.state = idx.x + params.width * idx.y;
  for (int i = 0; i < 10; ++i) { frandom(payload.rng); }

  for (int idx_sample = 0; idx_sample < params.n_samples; ++idx_sample) {
    // generate initial ray from camera
    const float2 uv = make_float2((2.0f * idx.x - dim.x) / dim.x,
                                  (2.0f * idx.y - dim.y) / dim.y);
    float3 ray_origin, ray_direction;
    sample_ray_pinhole_camera(uv, ray_origin, ray_direction);

    // start ray tracing from the camera
    payload.throughput = make_float3(1);
    payload.origin = ray_origin;
    payload.direction = ray_direction;
    payload.done = false;
    for (int depth = 0; depth < 3; ++depth) {
      trace_radiance(params.gas_handle, ray_origin, ray_direction, 0.0f, 1e9f,
                     &payload);

      if (payload.done) { break; }

      // advance ray
      ray_origin = payload.origin;
      ray_direction = payload.direction;
    }

    // accumulate contribution
    radiance += payload.throughput;
  }

  // take average
  radiance /= params.n_samples;

  // write radiance to frame buffer
  params.framebuffer[idx.x + params.width * idx.y] =
      make_float4(radiance, 1.0f);
}

extern "C" __global__ void __miss__radiance()
{
  const MissSbtRecordData* sbt =
      reinterpret_cast<MissSbtRecordData*>(optixGetSbtDataPointer());

  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  payload->throughput *= sbt->bg_color;
  payload->done = true;
}

extern "C" __global__ void __miss__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visibility = 1.0f;
}

extern "C" __global__ void __closesthit__radiance()
{
  RadiancePayload* payload = get_payload_ptr<RadiancePayload>();
  const HitGroupSbtRecordData* sbt =
      reinterpret_cast<HitGroupSbtRecordData*>(optixGetSbtDataPointer());

  // compute face normal
  // TODO: remove this calculation, store normals(ptr) in SBT
  const int prim_idx = optixGetPrimitiveIndex();
  const float3 v0 = sbt->vertices[3 * prim_idx + 0];
  const float3 v1 = sbt->vertices[3 * prim_idx + 1];
  const float3 v2 = sbt->vertices[3 * prim_idx + 2];
  const float3 n = normalize(cross(v1 - v0, v2 - v0));

  // compute tangent space basis
  float3 t, b;
  orthonormal_basis(n, t, b);

  // sample shadow ray direction
  const float3 wi = sample_cosine_weighted_hemisphere(frandom(payload->rng),
                                                      frandom(payload->rng));
  const float3 wi_world = local_to_world(wi, t, n, b);

  // trace shadow ray
  const float3 shadow_ray_origin =
      optixGetWorldRayOrigin() +
      optixGetRayTmax() * optixGetWorldRayDirection() + RAY_EPS * n;
  const float3 shadow_ray_direction = wi_world;
  ShadowPayload shadow_payload;
  trace_shadow(params.gas_handle, shadow_ray_origin, shadow_ray_direction, 0.0f,
               1e9f, &shadow_payload);

  // multiply visibility
  payload->throughput *= shadow_payload.visibility * sbt->material.base_color;
  payload->done = true;
}

extern "C" __global__ void __closesthit__shadow()
{
  ShadowPayload* payload = get_payload_ptr<ShadowPayload>();
  payload->visibility = 0.0f;
}