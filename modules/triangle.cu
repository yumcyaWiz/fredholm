#include "hip/hip_runtime.h"
#include <optix.h>

#include "shared.h"
#include "sutil/vec_math.h"

extern "C" {
__constant__ Params params;
}

static __forceinline__ __device__ void set_payload(float3 p)
{
  optixSetPayload_0(__float_as_int(p.x));
  optixSetPayload_1(__float_as_int(p.y));
  optixSetPayload_2(__float_as_int(p.z));
}

static __forceinline__ __device__ void sample_ray_pinhole_camera(
    float2 uv, float3& origin, float3& direction)
{
  const float3 p_sensor =
      params.cam_origin + uv.x * params.cam_right + uv.y * params.cam_up;
  const float3 p_pinhole = params.cam_origin + params.cam_forward;

  origin = params.cam_origin;
  direction = normalize(p_pinhole - p_sensor);
}

extern "C" __global__ void __raygen__rg()
{
  const uint3 idx = optixGetLaunchIndex();
  const uint3 dim = optixGetLaunchDimensions();

  const float2 uv = make_float2((2.0f * idx.x - dim.x) / dim.x,
                                (2.0f * idx.y - dim.y) / dim.y);
  float3 ray_origin, ray_direction;
  sample_ray_pinhole_camera(uv, ray_origin, ray_direction);

  unsigned int p0, p1, p2;
  optixTrace(params.handle, ray_origin, ray_direction, 0.0f, 1e9f, 0.0f,
             OptixVisibilityMask(255), OPTIX_RAY_FLAG_NONE, 0, 1, 0, p0, p1,
             p2);
  float4 color;
  color.x = __int_as_float(p0);
  color.y = __int_as_float(p1);
  color.z = __int_as_float(p2);

  params.image[idx.x + idx.y * params.image_width] = color;
}

extern "C" __global__ void __miss__ms() {}

extern "C" __global__ void __closesthit__ch()
{
  const float2 barycentrics = optixGetTriangleBarycentrics();

  set_payload(make_float3(barycentrics, 1.0f));
}